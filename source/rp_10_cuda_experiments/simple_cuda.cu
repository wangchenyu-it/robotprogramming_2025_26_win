#include "hip/hip_runtime.h"
#include <vector>
#include "cuda_common.h"
#include <iostream>

using namespace std;

void addArrayCPU(float* dest, const float* src1,  const float* src2, int size) {
  for (int i=0; i<size; ++i) {
    dest[i]=src1[i]+src2[i];
  }
}

__global__ void addArrayCUDA_kernel(float* dest, const float* src, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >=size)
      return;
    dest[tid]+=src[tid];
}

void addArrayCUDA(float* dest, const float* src1,  const float* src2, int size) {

  // copu on gpu
  float* ws_gpu;
  CUDA_CHECK(hipMalloc((void**)&ws_gpu, sizeof(float)*2*size));
  CUDA_CHECK(hipMemcpy(ws_gpu, src1, sizeof(float)*size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(ws_gpu+size, src2, sizeof(float)*size, hipMemcpyHostToDevice));
  
  int n_threads=1024;
  int n_blocks  = roundUp(size,n_threads);
  addArrayCUDA_kernel<<<n_blocks, n_threads>>>(ws_gpu, ws_gpu+size, size);
  CUDA_CHECK(hipMemcpy(dest, ws_gpu, sizeof(float)*size, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(ws_gpu));
}



float dotProductArrayCPU(const float* src1,  const float* src2, int size) {
  float result=0;
  for (int i=0; i<size; ++i) {
    result+=src1[i]*src2[i];
  }
  return result;
}


__global__ void mulArrayCUDA_kernel(float* dest, const float* src, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >=size)
      return;
    dest[tid]*=src[tid];
}

__global__ void reduceArrayCUDA_kernel(float* dest, const float* src, int dest_size, int src_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >=dest_size)
      return;
    int dest_idx=tid;
    int src_idx=tid*2;
    if (src_idx>=src_size)
      return;
    dest[dest_idx]=src[src_idx];
    ++src_idx;
    if (src_idx>=src_size)
      return;
    dest[dest_idx]+=src[src_idx];
}

float dotProductArrayCUDA(const float* src1,  const float* src2, int size) {
  if (! size)
    return 0;
  
  float* ws=0;
  CUDA_CHECK(hipMalloc((void**)&ws, sizeof(float)*2*size));
  float* prod_ws=ws;
  float* red_ws=ws+size;
  CUDA_CHECK(hipMemcpy(prod_ws, src1, sizeof(float)*size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(red_ws,  src2, sizeof(float)*size, hipMemcpyHostToDevice));

  // a*b
  int n_threads=1024;
  int n_blocks  = roundUp(size,n_threads);
  mulArrayCUDA_kernel<<<n_blocks, n_threads>>>(prod_ws, red_ws, size);

  // reduction
  int src_size=size;
  float* src=prod_ws;
  float* dest=red_ws;
  while (src_size>1) {
    int dest_size=roundUp(src_size, 2);
    int n_threads=1024;
    int n_blocks  = roundUp(size,n_threads);
    reduceArrayCUDA_kernel<<<n_blocks, n_threads>>>(dest, src, dest_size, src_size);
    std::swap(src, dest);
    src_size=dest_size;
  }
  float result;
  CUDA_CHECK(hipMemcpy(&result, src, sizeof(float), hipMemcpyDeviceToHost));
  return result;
}

std::ostream& operator<<(std::ostream& os, const std::vector<float>& src) {
  for (const auto& v: src)
    os << v << " ";
  return os;
}


int main(int argc, char** argv) {
  int size=20;
  std::vector<float> src1(size), src2(size);
  for (auto& v: src1)
    v=drand48();

  cerr << "src1: " << src1 << endl;
  for (auto& v: src2)
    v=drand48();
  cerr << "src2: " << src2 << endl;

  cerr << "sum (cpu)";
  std::vector<float> dest(size);
  addArrayCPU(&dest[0], &src1[0], &src2[0], size);
  cerr << dest << endl;

  cerr << "sum (cuda)";
  addArrayCUDA(&dest[0], &src1[0], &src2[0], size);
  cerr << dest << endl;

  cerr << "dot (cpu)" << dotProductArrayCPU(&src1[0], &src2[0], size);
  cerr << "dot (cuda)" << dotProductArrayCUDA(&src1[0], &src2[0], size);
  
}
